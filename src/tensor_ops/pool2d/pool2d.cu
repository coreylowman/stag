
#include <hip/hip_runtime.h>
struct Pool2dOp {
    size_t kernel;
    size_t stride;
    size_t padding;
    size_t batch;
    size_t chan;
    size_t h_in;
    size_t h_out;
    size_t w_in;
    size_t w_out;
};

template<typename T>
__device__ void avg_pool2d_forward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;
    
    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp += inp[inp_i];
        }
    }

    tmp /= static_cast<T>(op.kernel * op.kernel);
    out[i] = tmp;
}

template<typename T>
__device__ void avg_pool2d_backward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *grad_inp,
    const T *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const T *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];
            tmp += grad_out[out_i];
        }
    }

    grad_inp[i] += tmp / static_cast<T>(op.kernel * op.kernel);
}

__device__ __forceinline__ float maxNonAtomic(float a, float b) {
    return fmaxf(a, b);
}

__device__ __forceinline__ double maxNonAtomic(double a, double b) {
    return fmax(a, b);
}

template<typename T>
__device__ void max_pool2d_forward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    T tmp = -INFINITY;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp = maxNonAtomic(tmp, inp[inp_i]);
        }
    }

    out[i] = tmp;
}

template<typename T>
__device__ void max_pool2d_backward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *grad_inp,
    const T *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const T *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    const T inp_v = inp[i];

    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];

            if (out[out_i] == inp_v) {
                tmp += grad_out[out_i];
            }
        }
    }

    grad_inp[i] += tmp;
}

__device__ __forceinline__ float minNonAtomic(float a, float b) {
    return fminf(a, b);
}

__device__ __forceinline__ double minNonAtomic(double a, double b) {
    return fmin(a, b);
}

template<typename T>
__device__ void min_pool2d_forward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *out // 4d (Batch, Channels, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_out * op.w_out;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    T tmp = INFINITY;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            const size_t y_plus_p = oh * op.stride + k1;
            if (y_plus_p < op.padding) { continue; }
            const size_t y = y_plus_p - op.padding;
            if (y >= op.h_in) { continue; }
            const size_t x_plus_p = ow * op.stride + k2;
            if (x_plus_p < op.padding) { continue; }
            const size_t x = x_plus_p - op.padding;
            if (x >= op.w_in) { continue; }

            auto inp_i = b * inp_strides[0] + c * inp_strides[1] + y * inp_strides[2] + x * inp_strides[3];
            tmp = minNonAtomic(tmp, inp[inp_i]);
        }
    }

    out[i] = tmp;
}

template<typename T>
__device__ void min_pool2d_backward(
    const Pool2dOp op,
    const size_t *inp_strides,
    const size_t *out_strides,
    const T *inp, // 4d (Batch, Channels, Height, Width)
    T *grad_inp,
    const T *out, // 4d (Batch, Channels, HeightOut, WidthOut)
    const T *grad_out
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t numel = op.batch * op.chan * op.h_in * op.w_in;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t c = idx % op.chan;
    idx /= op.chan;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    const T inp_v = inp[i];

    T tmp = 0.0;
    for(size_t k1 = 0; k1 < op.kernel; k1++) {
        for (size_t k2 = 0; k2 < op.kernel; k2++) {
            size_t oh = y + op.padding;
            if (oh < k1) { continue; }
            oh -= k1;
            if (oh % op.stride != 0) { continue; }
            oh /= op.stride;
            if (oh >= op.h_out) { continue; }

            size_t ow = x + op.padding;
            if (ow < k2) { continue; }
            ow -= k2;
            if (ow % op.stride != 0) { continue; }
            ow /= op.stride;
            if (ow >= op.w_out) { continue; }

            auto out_i = b * out_strides[0] + c * out_strides[1] + oh * out_strides[2] + ow * out_strides[3];

            if (out[out_i] == inp_v) {
                tmp += grad_out[out_i];
            }
        }
    }

    grad_inp[i] += tmp;
}

#define POOL_OP(TYPENAME, FORWARD, BACKWARD, FORWARD_FN, BACKWARD_FN) \
extern "C" __global__ void FORWARD( \
    const Pool2dOp op, \
    const size_t *inp_strides, \
    const size_t *out_strides, \
    const TYPENAME *inp, \
    TYPENAME *out \
) { \
    FORWARD_FN(op, inp_strides, out_strides, inp, out); \
} \
extern "C" __global__ void BACKWARD( \
    const Pool2dOp op, \
    const size_t *inp_strides, \
    const size_t *out_strides, \
    const TYPENAME *inp, \
    TYPENAME *grad_inp, \
    const TYPENAME *out, \
    const TYPENAME *grad_out \
) { \
    BACKWARD_FN(op, inp_strides, out_strides, inp, grad_inp, out, grad_out); \
}

POOL_OP(
    float,
    avg_pool2d_forward_f32, avg_pool2d_backward_f32,
    avg_pool2d_forward, avg_pool2d_backward
);
POOL_OP(
    float,
    min_pool2d_forward_f32, min_pool2d_backward_f32,
    min_pool2d_forward, min_pool2d_backward
);
POOL_OP(
    float,
    max_pool2d_forward_f32, max_pool2d_backward_f32,
    max_pool2d_forward, max_pool2d_backward
);

POOL_OP(
    double,
    avg_pool2d_forward_f64, avg_pool2d_backward_f64,
    avg_pool2d_forward, avg_pool2d_backward
);
POOL_OP(
    double,
    min_pool2d_forward_f64, min_pool2d_backward_f64,
    min_pool2d_forward, min_pool2d_backward
);
POOL_OP(
    double,
    max_pool2d_forward_f64, max_pool2d_backward_f64,
    max_pool2d_forward, max_pool2d_backward
);
